#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template void quickselect<float,select_config<10, 10, 8, false, true, true, 8, 9, 10>>(float* in, float* tmp, index* count_tmp, index size, index rank, float* out);
template __host__ __device__ void collect_bucket_indirect<double,select_config<10, 10, 8, true, false, true, 8, 10, 10>>(const double* data, const poracle* oracles_packed, const index* prefix_sum, double* out, index size, const oracle* bucket, index* atomic);
template __device__ __host__ void partition<double,select_config<10, 5, 8, false, true, true, 8, 10, 10>>(const double* in, double* out, index* counts, index size, double pivot);
template void sampleselect_multi<float,select_config<10, 10, 7, false, true, true, 8, 10, 10>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template void sampleselect_host<double,select_config<9, 10, 8, true, true, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __global__ void kernels::reduce_counts<select_config<10, 11, 9, false, true, false, 8, 10, 10>>(const index* in, index* out, index);
template void sampleselect_host<float,select_config<10, 10, 8, true, true, true, 4, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template __host__ __device__ void collect_bucket<float,select_config<10, 10, 8, true, true, false, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* prefix_sum, float* out, index size, oracle bucket, index* atomic);
template void sampleselect_multi<float,select_config<10, 10, 6, true, true, true, 8, 10, 10, false, 1024>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template void sampleselect_host<double,select_config<10, 10, 6, true, true, true, 8, 10, 10, false, 1024>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template void sampleselect_multi<float,select_config<10, 10, 8, false, true, true, 8, 8, 10>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template __global__ void kernels::count_buckets<double, select_config<10, 8, 6, true, true, false, 8, 10, 10>>(const double* in, const double* tree, index* counts, poracle* oracles, index size, index workcount);
}