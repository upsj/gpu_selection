#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __global__ void kernels::count_buckets<double, select_config<10, 9, 7, false, true, false, 8, 10, 10>>(const double* in, const double* tree, index* counts, poracle* oracles, index size, index workcount);
template void sampleselect<float,select_config<10, 10, 8, false, true, true, 8, 8, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect<double,select_config<10, 10, 8, false, true, true, 2, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __host__ __device__ void collect_bucket_indirect<float,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* prefix_sum, float* out, index size, const oracle* bucket, index* atomic);
template __global__ void kernels::count_buckets<float, select_config<10, 12, 10, false, true, false, 8, 10, 10>>(const float* in, const float* tree, index* counts, poracle* oracles, index size, index workcount);
template __global__ void kernels::partition_distr<float,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const float* in, float* out, const index* counts, index size, float pivot, index workcount);
template __host__ __device__ launch_parameters get_launch_parameters<float,select_config<10, 10, 8, true, true, false, 8, 10, 10>>(index size);
template void sampleselect_host<float,select_config<8, 10, 8, false, true, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template __host__ __device__ launch_parameters get_launch_parameters<double,select_config<10, 9, 7, true, true, false, 8, 10, 10>>(index size);
template __global__ void kernels::partition_distr<double,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const double* in, double* out, const index* counts, index size, double pivot, index workcount);
template __global__ void kernels::prefix_sum_counts<select_config<10, 12, 10, false, true, false, 8, 10, 10>>(index* in, index* out, index);
}