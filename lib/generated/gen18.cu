#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __host__ __device__ void ssss_merged<float,select_config<10, 10, 8, false, false, true, 8, 10, 10>>(const float* in, float* out, poracle* oracles, index offset, const index* ranks, index rank_offset, index rank_base, const kernels::ssss_multi_aux<float, select_config<10, 10, 8, false, false, true, 8, 10, 10>>* aux_in, kernels::ssss_multi_aux<float, select_config<10, 10, 8, false, false, true, 8, 10, 10>>* aux_outs, float* out_tree);
template void quickselect<float,select_config<10, 10, 8, true, true, true, 2, 10, 10>>(float* in, float* tmp, index* count_tmp, index size, index rank, float* out);
template __global__ void kernels::reduce_counts<select_config<10, 11, 9, true, true, false, 8, 10, 10>>(const index* in, index* out, index);
template __global__ void kernels::count_buckets<float, select_config<10, 9, 7, false, true, false, 8, 10, 10>>(const float* in, const float* tree, index* counts, poracle* oracles, index size, index workcount);
template void sampleselect_multi<float,select_config<10, 10, 8, false, true, true, 8, 10, 10, false, 8, 0>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template __global__ void kernels::count_buckets<double, select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const double* in, const double* tree, index* counts, poracle* oracles, index size, index workcount);
template void sampleselect_host<double,select_config<10, 10, 6, true, false, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __global__ void partition_count<float,select_config<10, 5, 8, true, true, true, 8, 10, 10>>(const float* in, index* counts, index size, float pivot, index workcount);
template __global__ void kernels::prefix_sum_counts<select_config<10, 10, 8, false, true, false, 8, 10, 10>>(index* in, index* out, index);
template __host__ __device__ void collect_bucket<double,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const double* data, const poracle* oracles_packed, const index* prefix_sum, double* out, index size, oracle bucket, index* atomic);
template void sampleselect_host<double,select_config<10, 10, 8, false, true, true, 2, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
}