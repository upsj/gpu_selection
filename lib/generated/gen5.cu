#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __host__ __device__ void collect_bucket<float,select_config<10, 12, 10, true, true, false, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* prefix_sum, float* out, index size, oracle bucket, index* atomic);
template __global__ void kernels::prefix_sum_counts<select_config<10, 10, 8, true, true, false, 8, 10, 10>>(index* in, index* out, index);
template __global__ void kernels::count_buckets<double, select_config<10, 10, 8, true, false, false, 8, 10, 10>>(const double* in, const double* tree, index* counts, poracle* oracles, index size, index workcount);
template __host__ __device__ launch_parameters get_launch_parameters<float,select_config<10, 11, 9, false, true, false, 8, 10, 10>>(index size);
template __global__ void partition_count<float,select_config<10, 5, 8, false, true, true, 8, 10, 10>>(const float* in, index* counts, index size, float pivot, index workcount);
template void sampleselect_multi<double,select_config<10, 10, 8, false, true, true, 8, 9, 10>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
template __host__ __device__ void collect_bucket<double,select_config<10, 10, 8, true, false, false, 8, 10, 10>>(const double* data, const poracle* oracles_packed, const index* prefix_sum, double* out, index size, oracle bucket, index* atomic);
template __global__ void kernels::partition_distr<float,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(const float* in, float* out, const index* counts, index size, float pivot, index workcount);
template __global__ void kernels::count_buckets<float, select_config<10, 12, 10, true, true, false, 8, 10, 10>>(const float* in, const float* tree, index* counts, poracle* oracles, index size, index workcount);
template void sampleselect<double,select_config<10, 10, 8, false, true, true, 8, 10, 10, false, 4>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __host__ __device__ void collect_bucket<float,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* prefix_sum, float* out, index size, oracle bucket, index* atomic);
template __global__ void kernels::partition<double,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(const double* in, double* out, index* atomic, index size, double pivot, index workcount);
}