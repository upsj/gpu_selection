#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __host__ __device__ void collect_bucket<double,select_config<10, 9, 7, true, true, false, 8, 10, 10>>(const double* data, const poracle* oracles_packed, const index* prefix_sum, double* out, index size, oracle bucket, index* atomic);
template void sampleselect_host<float,select_config<10, 10, 8, true, true, true, 8, 10, 10, true>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect_host<double,select_config<10, 10, 8, true, true, true, 4, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __host__ __device__ void collect_buckets<float,select_config<10, 10, 8, false, false, true, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* block_prefix_sum, const index* bucket_out_ranges, float* out, index size, mask* buckets, index* atomic);
template __device__ void kernels::masked_prefix_sum<7>(index* counts, const mask* m);
template void quickselect<double,select_config<9, 10, 8, true, true, true, 8, 10, 10>>(double* in, double* tmp, index* count_tmp, index size, index rank, double* out);
template void sampleselect_multi<double,select_config<9, 10, 8, true, true, true, 8, 10, 10>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
template __global__ void kernels::prefix_sum_counts<select_config<10, 12, 10, true, true, false, 8, 10, 10>>(index* in, index* out, index);
template void sampleselect<float,select_config<9, 10, 8, true, true, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void quickselect<float,select_config<8, 10, 8, false, true, true, 8, 10, 10>>(float* in, float* tmp, index* count_tmp, index size, index rank, float* out);
template __host__ __device__ void collect_bucket_indirect<double,select_config<10, 10, 8, false, false, true, 8, 10, 10>>(const double* data, const poracle* oracles_packed, const index* prefix_sum, double* out, index size, const oracle* bucket, index* atomic);
}