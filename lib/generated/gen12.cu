#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __global__ void kernels::count_buckets<float, select_config<10, 9, 7, true, true, false, 8, 10, 10>>(const float* in, const float* tree, index* counts, poracle* oracles, index size, index workcount);
template void sampleselect<double,select_config<10, 10, 8, false, true, true, 8, 9, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __host__ __device__ void collect_bucket<double,select_config<10, 9, 7, false, true, false, 8, 10, 10>>(const double* data, const poracle* oracles_packed, const index* prefix_sum, double* out, index size, oracle bucket, index* atomic);
template __global__ void kernels::reduce_counts<select_config<10, 10, 8, false, true, false, 8, 10, 10>>(const index* in, index* out, index);
template void quickselect<float,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(float* in, float* tmp, index* count_tmp, index size, index rank, float* out);
template void sampleselect_host<float,select_config<10, 10, 7, false, true, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect_multi<double,select_config<10, 10, 8, true, true, true, 2, 10, 10>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
template void sampleselect_multi<float,select_config<10, 10, 8, false, true, true, 8, 10, 10, false, 4>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template __host__ __device__ void build_searchtree<float,select_config<10, 10, 8, true, true, false, 8, 10, 10>>(const float* in, float* out, index size);
template void quickselect_multi<double,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(double* in, double* tmp, index* count_tmp, index size, const index* ranks, index rank_count, double* out);
template void sampleselect<float,select_config<10, 10, 6, false, true, true, 8, 10, 10, false, 1024>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
}