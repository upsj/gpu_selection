#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __global__ void kernels::partition_distr<float,select_config<10, 5, 8, true, true, true, 8, 10, 10>>(const float* in, float* out, const index* counts, index size, float pivot, index workcount);
template void quickselect_multi<float,select_config<10, 5, 8, false, true, true, 8, 10, 10>>(float* in, float* tmp, index* count_tmp, index size, const index* ranks, index rank_count, float* out);
template __global__ void kernels::partition<double,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const double* in, double* out, index* atomic, index size, double pivot, index workcount);
template void sampleselect_multi<double,select_config<8, 10, 8, false, true, true, 8, 10, 10>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
template __host__ __device__ void collect_bucket<float,select_config<10, 9, 7, false, true, false, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* prefix_sum, float* out, index size, oracle bucket, index* atomic);
template __host__ __device__ void collect_bucket_indirect<float,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* prefix_sum, float* out, index size, const oracle* bucket, index* atomic);
template __global__ void kernels::reduce_counts<select_config<10, 10, 8, false, true, true, 8, 10, 10>>(const index* in, index* out, index);
template void sampleselect_multi<double,select_config<10, 10, 8, false, true, true, 8, 10, 10, false, 8, 0>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
template void sampleselect_host<float,select_config<10, 10, 8, false, false, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void quickselect<float,select_config<8, 10, 8, true, true, true, 8, 10, 10>>(float* in, float* tmp, index* count_tmp, index size, index rank, float* out);
template __global__ void kernels::count_buckets<double, select_config<10, 11, 9, true, true, false, 8, 10, 10>>(const double* in, const double* tree, index* counts, poracle* oracles, index size, index workcount);
}