#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __global__ void kernels::count_buckets<float, select_config<10, 8, 6, false, true, false, 8, 10, 10>>(const float* in, const float* tree, index* counts, poracle* oracles, index size, index workcount);
template void sampleselect_host<float,select_config<10, 10, 8, false, true, true, 8, 8, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect_host<float,select_config<8, 10, 8, true, true, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect_multi<float,select_config<10, 10, 8, false, true, true, 8, 10, 10, false, 16>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template void sampleselect<float,select_config<10, 10, 8, false, true, true, 8, 9, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void quickselect<float,select_config<10, 10, 8, false, true, true, 2, 10, 10>>(float* in, float* tmp, index* count_tmp, index size, index rank, float* out);
template __global__ void kernels::count_buckets<float, select_config<10, 11, 9, false, true, false, 8, 10, 10>>(const float* in, const float* tree, index* counts, poracle* oracles, index size, index workcount);
template void sampleselect<double,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template void sampleselect_host<double,select_config<10, 10, 8, false, true, true, 8, 10, 10, true>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __host__ __device__ void collect_buckets<float,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* block_prefix_sum, const index* bucket_out_ranges, float* out, index size, mask* buckets, index* atomic);
template __host__ __device__ void collect_bucket<double,select_config<10, 8, 6, false, true, false, 8, 10, 10>>(const double* data, const poracle* oracles_packed, const index* prefix_sum, double* out, index size, oracle bucket, index* atomic);
template __host__ __device__ void build_searchtree<float,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(const float* in, float* out, index size);
}