#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template void sampleselect_multi<float,select_config<10, 10, 8, false, true, true, 4, 10, 10>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template void sampleselect_multi<double,select_config<10, 10, 8, false, false, true, 8, 10, 10>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
template __host__ __device__ launch_parameters get_launch_parameters<double,select_config<10, 10, 8, true, true, false, 8, 10, 10>>(index size);
template __host__ __device__ void build_searchtree<float,select_config<10, 10, 8, false, false, false, 8, 10, 10>>(const float* in, float* out, index size);
template __global__ void kernels::partition<double,select_config<10, 5, 8, false, true, true, 8, 10, 10>>(const double* in, double* out, index* atomic, index size, double pivot, index workcount);
template void sampleselect<float,select_config<10, 10, 6, true, false, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect<double,select_config<10, 10, 6, true, true, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template void quickselect<double,select_config<10, 5, 8, true, true, true, 8, 10, 10>>(double* in, double* tmp, index* count_tmp, index size, index rank, double* out);
template void quickselect<double,select_config<10, 10, 8, false, true, true, 8, 9, 10>>(double* in, double* tmp, index* count_tmp, index size, index rank, double* out);
template __global__ void kernels::prefix_sum_counts<select_config<10, 10, 8, true, true, true, 8, 10, 10>>(index* in, index* out, index);
template void sampleselect_multi<float,select_config<10, 10, 8, true, true, true, 8, 10, 10, false, 16>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
}