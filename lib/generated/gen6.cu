#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __host__ __device__ void build_searchtree<float,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const float* in, float* out, index size);
template void sampleselect<double,select_config<10, 10, 8, false, false, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __global__ void kernels::prefix_sum_counts<select_config<10, 9, 7, true, true, false, 8, 10, 10>>(index* in, index* out, index);
template __host__ __device__ void count_buckets<float,select_config<10, 10, 8, true, true, false, 8, 10, 10>>(const float* in, const float* tree, index* localcounts, index* counts, poracle* oracles, index size);
template void sampleselect_multi<float,select_config<10, 10, 8, true, false, true, 8, 10, 10>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template void quickselect_multi<double,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(double* in, double* tmp, index* count_tmp, index size, const index* ranks, index rank_count, double* out);
template __host__ __device__ void build_searchtree<float,select_config<10, 8, 6, false, true, false, 8, 10, 10>>(const float* in, float* out, index size);
template void sampleselect<float,select_config<10, 10, 8, false, true, true, 4, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect<double,select_config<10, 10, 6, true, true, true, 8, 10, 10, false, 1024>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template void sampleselect_host<float,select_config<10, 10, 6, true, false, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect_multi<double,select_config<10, 10, 6, false, true, true, 8, 10, 10, false, 1024>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
template void sampleselect_host<float,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
}