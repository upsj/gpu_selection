#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template void sampleselect<double,select_config<10, 10, 8, true, true, true, 2, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template void sampleselect_host<double,select_config<10, 10, 8, false, true, true, 8, 8, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template void sampleselect_host<float,select_config<10, 10, 8, false, true, true, 8, 10, 10, true>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect_multi<float,select_config<10, 10, 8, true, true, true, 8, 10, 10, true>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template __global__ void partition_count<double,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(const double* in, index* counts, index size, double pivot, index workcount);
template __global__ void kernels::reduce_counts<select_config<10, 8, 6, false, true, false, 8, 10, 10>>(const index* in, index* out, index);
template void sampleselect<double,select_config<10, 10, 7, false, true, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template void sampleselect_host<float,select_config<10, 10, 8, false, true, true, 8, 10, 10, false, 8, 0>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect<float,select_config<10, 10, 6, false, false, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void quickselect<double,select_config<10, 5, 8, false, true, true, 8, 10, 10>>(double* in, double* tmp, index* count_tmp, index size, index rank, double* out);
template __host__ __device__ void build_searchtree<float,select_config<10, 12, 10, true, true, false, 8, 10, 10>>(const float* in, float* out, index size);
}